#include "hip/hip_runtime.h"

#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include "hip/hip_runtime.h"
#include "hiprand.h"
#include "hipblas.h"

extern "C"
{
#include "sgemm.h"
#include "hip/hip_runtime.h"
#include "utils.h"
#include "gemm.h"
}


__global__ void sgemm(int TA, int TB, int M, int N, int K, float ALPHA, 
        float *A_gpu, int lda, 
        float *B_gpu, int ldb,
        float BETA,
        float *C_gpu, int ldc)
	{
















		//printf("Kernel call has executed for gemm\n");
				
    		int row = blockDim.y * blockIdx.y + threadIdx.y;
    		int col = blockDim.x * blockIdx.x + threadIdx.x;
		//printf("A is %f\n",A_gpu[42]);			Will print multiple values as multiple threads

   		if (row > M || col > N) return; // Check for k as well, gotta use lda,ldb as well.
  
   			double prod = 0;
			int kk;
			for (kk = 0; kk < N; ++kk){
			    prod += A_gpu[row * lda + kk] * B_gpu[kk * ldb + col];
			    //printf("%d\n",prod);
			   }
			C_gpu[row*ldc + col] = ALPHA * prod + BETA * C_gpu[row*ldc+col];    
			printf("Kernel call has completed for gemm\n");		

	}


void sgemm_gpu(int TA, int TB, int M, int N, int K, float ALPHA, 
        float *A_gpu, int lda, 
        float *B_gpu, int ldb,
        float BETA,
        float *C_gpu, int ldc)
{
    //printf("Cublas has started Successfully\n");
    //printf("Printing out the parameters\n");
  //printf("Gpu: %d %d %d %d %d %f %d %d %f %d\n",TA, TB, M, N, K, ALPHA, lda, ldb, BETA, ldc);

    printf("These are the calls to gemm gpu\n");
    const dim3 blocksize(32,16);
    const dim3 gridsize(N/blocksize.y +1,M/blocksize.x+1);
    sgemm<<<gridsize,blocksize>>>(TA, TB, M, N, K, ALPHA, 
        A_gpu, lda, 
        B_gpu, ldb,
        BETA,
        C_gpu, ldc);


    check_error(hipPeekAtLastError());
  //printf("Cublas has ended Successfully\n");
}




















